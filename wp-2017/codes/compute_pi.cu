/*-------------------------------------------------------------------------
!    THIS SOFTWARE IS PROVIDED "AS IS", AND JAYANTI PRASAD  MAKE NO
     REPRESENTATIONS OR WARRANTIES, EXPRESS OR IMPlIED.
     By way of example, but not limitation,
     JAYANTI PRASAD MAKE NO REPRESENTATIONS OR WARRANTIES OF
     MERCHANTABIlITY OR FITNESS FOR ANY PARTICUlAR PURPOSE OR THAT
     THE USE OF THE lICENSED SOFTWARE OR DOCUMENTATION WIll NOT INFRINGE
     ANY THIRD PARTY PATENTS, COPYRIGHTS, TRADEMARKS OR OTHER RIGHTS.
-------------------------------------------------------------------*/

/*--------------------------- Parallel Programming with CUDA : Demo 1 ------------->>>

 This program computes the value of pi by integrating 1/(1+x^2) between limits [0,1]
 and demonsrate the use of cuda.

 In the present example cuda threads are in the form of a three dimensional block  
 of size 4 X 4 X 4 and which a member of 1 grid of size 1 X 1 X 1.

 Total number of cuda threads (nthreads) = number of grids X number of threads in a block. 

 User has to specify the number of point (npoints) used for the integartion at run time. 

 This program contains two parts : one compute (which runs on the device i.e., GPU, 
 and another which runs on the host i.e., the CPU.

 The number of compuations done by a thread = npoints/nthreads

 Each thread computes a local sum and finally all local sums are added in the local sum of
 the thread with id 0 


Important CUDA concepts used in the program:

 1. cudaMalloc : used to assign memory on the device i.e., GPU 
 2. cudaMemcpy : used to copy the value of a variable from the GPU memory space to CPU 
 3. dim3 : this is type of grid and blocks and can be used to specify the dimensionality 
 4. cuda calling fasion:

    compute <<<dimGrid, dimBlock>>>(local_sum,nval,BLOCK_SIZE);

  by using <<< >>> we can tell the GPU that how many grid or blocks should be used 
  to arrange cuda threads.


CUDA Philosophy:
  People who are familer with shared memory programming (pthread,opneMP) can recall that 
  there is just one memory space which can shared by all the threads. In case of CUDA 
  there are two memory spaces, one for CPU and another for GPU. Programmer has to copy the 
  values of variables from one space to another using cudaMemcpy function.
 

 I think this should be good enough for you to get strated with cuda ! 

 You can talk to me if you are futher interested in learning cuda like me. 

                              Jayanti Prasad (prasad.jaaynti@gmail.com)
                              Jan 21, 2010  
<<<-------------------------------------------------------------------------------------*/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 4 

__global__ void compute(float *local_sum, float nval, int threadDim){
  
  int i,tidx,tidy,tidz,myid,nthreads;   
  float x,sum,h,tmp;
  
  tidx = threadIdx.x;
  tidy = threadIdx.y;
  tidz = threadIdx.z;
  
  myid = tidz + threadDim*(tidy+threadDim*tidx);
  nthreads = threadDim * threadDim * threadDim;
  
  h = 1.0/nval; 
  sum = 0.0;
  x = 0.0;
  
  local_sum[myid] = 0.0;
  
  for(i= myid + 1; i  <= nval; i+= nthreads)
    {
      x = h * ( (float)i - 0.5 );
      sum += (4.0 /(1.0 + x*x));
    }
  
  local_sum[myid] = h * sum;
  
  __syncthreads();
  
  if( myid == 0 )
    {
      tmp = 0.0f;
      for(i = 0; i  < nthreads;i++)
	tmp  += local_sum[i];
      local_sum[0] = tmp;
    }
}
// this was the device part 

int main (int argc, char *argv[])
{
  float *local_sum; // for devive return variable 
  float *global_sum; // this is for the host (CPU)
  int   nval; // this is for device 
  int   numThread = BLOCK_SIZE*BLOCK_SIZE;
  
  nval = atoi(argv[1]); 
  
  global_sum  = (float*) malloc( sizeof(float));

  hipMalloc( (void**)&local_sum, numThread * sizeof(float));
  
  dim3 dimGrid(1,1,1);  
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE,BLOCK_SIZE);
  
  compute <<<dimGrid, dimBlock>>>(local_sum,nval,BLOCK_SIZE);
  
  hipMemcpy((void*)global_sum, (void*)local_sum, sizeof(float),hipMemcpyDeviceToHost );
  
  printf( "PI = %2.14f for %d # points\n", *global_sum,nval);
  
  hipFree(local_sum);
  
  free(global_sum);
  
  return(0); 
  
}
