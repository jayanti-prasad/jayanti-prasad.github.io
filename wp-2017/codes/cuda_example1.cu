/*------------------------------------------------------------------
                        CUDA Example 3
This program compute the sum of two vectrors of size N.
The work is distributed amoung cuda threads which are arranged in blocks.
Every block belongs to some grid and every thread belong to some block.
For simplicity I consider one dimensional grid only. However, blocks are
three dimenional. The number of blocks are computed on the basis of the
size of the blocks and the size of the problem.

number of blocks (dimensionality of 1 d grid)=probelem size (N)/
number of threads in a blocks. 

make sure you get an integer when divide your problem size by the number
of threads in a block. If you are not getting an integer, change the BLOCK_SIZE
accordingly.

Note that in any case you should not have more than 512 threads in a block.

This program has been tested on Nvidia Quadro FX 3700 and found to be working fine.
However, timing has not been checked.
                  -- Jayanti Prasad (prasad.jayanti@gmail.com)
                     Wed May  4 11:54:40 IST 2011 
-------------------------------------------------------------------*/
#include<stdio.h>
#include<hip/hip_runtime.h>

#define BLOCK_SIZE 8

__global__ void VecAdd(float *A, float *B, float *C, int N){

  int tid=threadIdx.x + blockDim.x*(threadIdx.y+blockDim.y * threadIdx.z); 
  int bid=blockIdx.x  + gridDim.x * blockIdx.y; 
 
  int nthreads = blockDim.x * blockDim.y * blockDim.z;
  int nblocks  = gridDim.x * gridDim.y; 

  int id      = tid + nthreads * bid;  
  int i;

  for(i=id; i < N; i+=nthreads*nblocks){
    C[i] = A[i] + B[i];
  }
   __syncthreads();
}

int main(int argc, char *argv[]){
  int i, N,grsz,blsz=BLOCK_SIZE;
  float *h_A,*h_B,*d_A,*d_B,*d_C,*h_C;

  if (argc < 2){
   fprintf(stderr,"./a.out  N [=512,1024 etc.]\n");
   return(-1);
  }

  N =  atoi(argv[1]);

  grsz = (int) N/(blsz*blsz*blsz); 

  printf("number grids = %d\n",grsz); 

  size_t size = N * sizeof(float);
  
  h_A = (float *)malloc(size);
  h_B = (float *)malloc(size);
  h_C = (float *)malloc(size);
  
  hipMalloc(&d_A,size);
  hipMalloc(&d_B,size);
  hipMalloc(&d_C,size);
  
  for(i=0; i < N; i++){
    h_A[i] = (float) i;
    h_B[i] = (float) i+1.0;
    h_C[i] = 0.0;
  }

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice); 
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  
  dim3 dimGrid(grsz); // 1 blocks
  dim3 dimBlock(blsz,blsz,blsz);  

  VecAdd<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
  
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  for(i=0; i < N; i++){
    fprintf(stdout,"A=%6.2f B=%6.2f A+B = %6.2f\n",h_A[i],h_B[i],h_C[i]);
  }
  hipFree(d_A); 
  hipFree(d_B);
  hipFree(d_C);
  
  return(0);
}
